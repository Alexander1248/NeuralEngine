
#include <hip/hip_runtime.h>
// Unary operations
extern "C"
__global__ void transpose(int width, int height,
            float* in, float* out) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height) {
        out[x + y * width] = in[y + x * height];
    }
}

extern "C"
__global__ void flipX(int width, int height,
            float* in, float* out) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height) {
        out[x + y * width] = in[(width - 1 - x) + y * width];
    }
}
extern "C"
__global__ void flipY(int width, int height,
            float* in, float* out) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height) {
        out[x + y * width] = in[x + (height - 1 - y) * width];
    }
}


extern "C"
__global__ void sortX(int width, int height, int iteration, int step,
            float* arr) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < (width >> 1) && y < height) {
        int blockSize = 1 << (iteration - step);
        int half = blockSize >> 1;
        int l = x % half;
        int g = x / half;    
        int pos1 = y * width + g * blockSize + l;
        int pos2;
        if (step == 0) pos2 = y * width + (g + 1) * blockSize - l - 1;
        else pos2 = y * width + g * blockSize + half + l;

        float buff = arr[pos1];
        arr[pos1] = arr[pos2];
        arr[pos2] = buff;
    }
}
extern "C"
__global__ void sortY(int width, int height, int iteration, int step,
            float* arr) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < (width >> 1) && y < height) {
       int blockSize = 1 << (iteration - step);
        int half = blockSize >> 1;
        int l = y % half;
        int g = y / half;    
        int pos1 = x + (l + g * blockSize) * width;
        int pos2;
        if (step == 0) pos2 = x + ((g + 1) * blockSize - l - 1) * width;
        else pos2 = x + (g * blockSize + half + l) * width;

        float buff = arr[pos1];
        arr[pos1] = arr[pos2];
        arr[pos2] = buff;
    }
}


extern "C"
__global__ void swapColumns(int width, int height, int c0, int c1,
            float* in, float* out) {
    int y = blockIdx.x * blockDim.x + threadIdx.x;

    if (y < height) {
        float value = in[c0 + y * width];
        out[c0 + y * width] = in[c1 + y * width];
        out[c1 + y * width] = value;
    }
}
extern "C"
__global__ void swapRows(int width, int r0, int r1,
            float* in, float* out) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;

    if (x < width) {
        float value = in[x + r0 * width];
        out[x + r0 * width] = in[x + r1 * width];
        out[x + r1 * width] = value;
    }
}

extern "C"
__global__ void rotate90(int width, int height,
            float* in, float* out) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height) {
        out[x + y * width] = in[(height - 1 - y) + x * height];
    }
}
extern "C"
__global__ void rotate180(int width, int height,
            float* in, float* out) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height) {
        out[x + y * width] = in[(width - 1 - x) + (height - 1 - y) * width];
    }
}
extern "C"
__global__ void rotate270(int width, int height,
            float* in, float* out) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height) {
        out[x + y * width] = in[y + (width - 1 - x) * height];
    }
}

extern "C"
__global__ void set(int width, int height, float value,
            float* out) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height) {
        out[x + y * width] = value;
    }
}

extern "C"
__global__ void sum(int size, float* mtx) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;

    if (x < size) {
        mtx[x] += mtx[x + size];
        mtx[x + size] = 0;
    }
}


// Binary operations
extern "C"
__global__ void tensorAdd(int width, int height,
            float* in1, float* in2, float* out) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height) {
        int pos = x + y * width;
        out[pos] = in1[pos] + in2[pos];
    }
}
extern "C"
__global__ void tensorSub(int width, int height,
            float* in1, float* in2, float* out) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height) {
        int pos = x + y * width;
        out[pos] = in1[pos] - in2[pos];
    }
}

extern "C"
 __global__ void concatenateVertical(int width, int height1, int height2,
             float* in1, float* in2, float* out) {
     int x = blockIdx.x * blockDim.x + threadIdx.x;
     int y = blockIdx.y * blockDim.y + threadIdx.y;

     if (x < width && y < height1 + height2) {
         int pos = x + y * width;

        if (y < height1) out[pos] = in1[pos];
        else {
            y -= height1;
            out[pos] = in2[x + y * width];
        }
     }
}

extern "C"
 __global__ void concatenateHorizontal(int width1, int width2, int height,
             float* in1, float* in2, float* out) {
     int x = blockIdx.x * blockDim.x + threadIdx.x;
     int y = blockIdx.y * blockDim.y + threadIdx.y;

     if (x < width1 + width2 && y < height) {
         int pos = x + y * (width1 + width2);

        if (x < width1) out[pos] = in1[x + y * width1];
        else {
            x -= width1;
            out[pos] = in2[x + y * width2];
        }
     }
}


extern "C"
__global__ void mul(int width, int height, float value,
            float* in, float* out) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height) {
        int pos = x + y * width;
        out[pos] = in[pos] * value;
    }
}
extern "C"
__global__ void tensorMul(int width, int height,
            float* in1, float* in2, float* out) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height) {
        int pos = x + y * width;
        out[pos] = in1[pos] * in2[pos];
    }
}
extern "C"
__global__ void matrixMul(int w1h2, int height1, int width2,
            float* in1, float* in2, float* out) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width2 && y < height1) {
        float sum = 0.0;
        for (int i = 0; i < w1h2; i++)
            sum += in1[i + y * w1h2] * in2[x + i * width2];

        out[x + y * width2] = sum;
    }
}

extern "C"
__global__ void tensorDiv(int width, int height,
            float* in1, float* in2, float* out) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height) {
        int pos = x + y * width;
        out[pos] = in1[pos] / in2[pos];
    }
}


extern "C"
__global__ void matrixConvEmptyBorder(int width, int height, int mx, int my,
            float* in, float* matrix, float* out) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height) {
        int hx = mx >> 1;
        int hy = my >> 1;

        float sum = 0;
        for (int dy = 0; dy < mx; dy++) {
            int py = y + dy - hy;
            if (py < 0) continue;
            if (py >= height) continue;

            for (int dx = 0; dx < mx; dx++) {
               int px = x + dx - hx;
                if (px < 0) continue;
                if (px >= width) continue;

                sum += in[px + py * width] * matrix[dx + dy * mx];
            }
        }
        out[x + y * width] = sum;
    }
}

extern "C"
__global__ void matrixConvExtendBorder(int width, int height, int mx, int my,
            float* in, float* matrix, float* out) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height) {
        int hx = mx >> 1;
        int hy = my >> 1;

        float sum = 0;
        for (int dy = 0; dy < mx; dy++) {
            int py = max(0, min(height - 1, y + dy - hy));

            for (int dx = 0; dx < mx; dx++) {
                int px = max(0, min(width - 1, x + dx - hx));

                sum += in[px + py * width] * matrix[dx + dy * mx];
            }
        }
        out[x + y * width] = sum;
    }
}

extern "C"
__global__ void matrixConvRepeatBorder(int width, int height, int mx, int my,
            float* in, float* matrix, float* out) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height) {
        int hx = mx >> 1;
        int hy = my >> 1;

        float sum = 0;
        for (int dy = 0; dy < mx; dy++) {
            int py = y + dy - hy;
            if (py < 0) py += height;
            if (py >= height) py -= height;

            for (int dx = 0; dx < mx; dx++) {
                int px = x + dx - hx;
                if (px < 0) px += width;
                if (px >= width) px -= width;

                sum += in[px + py * width] * matrix[dx + dy * mx];
            }
        }
        out[x + y * width] = sum;
    }
}

