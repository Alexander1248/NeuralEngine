
#include <hip/hip_runtime.h>
// Activation Functions
extern "C"
__global__ void relu(int width, int height, 
            double positiveCoefficient, double negativeCoefficient, 
            double* in, double* out) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height) {
        int pos = x + y * width;
        
        if (in[pos] >= 0) out[pos] = in[pos] * positiveCoefficient;
        else out[pos] = in[pos] * negativeCoefficient;
    }
}
extern "C"
__global__ void reluDer(int width, int height, 
            double positiveCoefficient, double negativeCoefficient, 
            double* in, double* out) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height) {
        int pos = x + y * width;
        
        if (in[pos] >= 0) out[pos] = positiveCoefficient;
        else out[pos] = negativeCoefficient;
    }
}


extern "C"
__global__ void sigmoid(int width, int height, 
            double force,
            double* in, double* out) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height) {
        int pos = x + y * width;
        
        out[pos] = 1.0 / (1.0 + exp(-force * in[pos]));
    }
}
extern "C"
__global__ void sigmoidDer(int width, int height, 
            double force,
            double* in, double* out) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height) {
        int pos = x + y * width;
        
        double val = exp(-force * in[pos]);
        double vp1 = 1.0 + val;
        out[pos] = force * val / (vp1 * vp1);
    }
}


extern "C"
__global__ void tangent(int width, int height, 
            double force,
            double* in, double* out) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height) {
        int pos = x + y * width;
        
        out[pos] = 2.0 / (1.0 + exp(-force * in[pos])) - 1;
    }
}
extern "C"
__global__ void tangentDer(int width, int height, 
            double force,
            double* in, double* out) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height) {
        int pos = x + y * width;

        double val = exp(-force * in[pos]);
        double vp1 = 1.0 + val;
        out[pos] = 2.0 * force * val / (vp1 * vp1);
    }
}


extern "C"
__global__ void softmax(int width, int height, 
            double force,
            double* in, double* out) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height) {
        int yw = y * width;
        int pos = x + yw;

        double sum = 0;
        for (int i = 0; i < width; i++)
            sum += exp(force * in[i + yw]);

        out[pos] = exp(force * in[pos]) / sum;
    }
}
extern "C"
__global__ void softmaxDer(int width, int height, 
            double force,
            double* in, double* out) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height) {
        int yw = y * width;
        int pos = x + yw;
        
        double sum = 0;
        for (int i = 0; i < width; i++)
            sum += exp(force * in[i + yw]);

        double e = exp(force * in[pos]);
        out[pos] =  force * e * (sum - e) / (sum * sum);
    }
}


extern "C"
__global__ void matrixMulBackpropagationErrorTraversal(int w1w2, int height1, int height2,
            double* currError, double* weigts, double* prevError) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < height2 && y < height1) {
        double sum = 0.0;
        for (int i = 0; i < w1w2; i++)
            sum += currError[i + y * height1] * weigts[x + i * w1w2];

        prevError[x + y * height2] = sum;
    }
}

extern "C"
__global__ void matrixMulBackpropagationWeightCorrection(int h1h2, int width1, int width2, double learningSpeed,
            double* input, double* error, double* weightsDelta) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width2 && y < width1) {
        double sum = 0.0;
        for (int i = 0; i < h1h2; i++)
            sum += input[y + i * width1] * error[x + i * width2];

        weightsDelta[x + y * width2] = sum;
    }
}



extern "C"
__global__ void matrixConvEmptyBorderBackpropagationErrorTraversal(int width, int height, int mx, int my,
            double* currError, double* matrix, double* prevError) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height) {
        int hx = mx >> 1;
        int hy = my >> 1;

        double sum = 0;
        for (int dy = 0; dy < mx; dy++) {
            int py = y + my - 1 - dy - hy;
            if (py < 0) continue;
            if (py >= height) continue;

            for (int dx = 0; dx < mx; dx++) {
               int px = x + mx - 1 - dx - hx;
                if (px < 0) continue;
                if (px >= width) continue;

                sum += currError[px + py * width] * matrix[dx + dy * width];
            }
        }
        prevError[x + y * width] = sum;
    }
}
extern "C"
__global__ void matrixConvEmptyBorderBackpropagationWeightCorrection(int width, int height, int mx, int my, double ls,
            double* input, double* error, double* matrixDelta) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < mx && y < my) {
        int hx = mx >> 1;
        int hy = my >> 1;

        int sx = x - hx;
        int sy = y - hy;

        double sum = 0;
        for (int dy = 0; dy < height; dy++) {
            int py = dy - sy;
            if (py < 0) continue;
            if (py >= height) continue;
            for (int dx = 0; dx < width; dx++) {
                int px = dx - sx;
                if (px < 0) continue;
                if (px >= width) continue;
                sum += error[dx + dy * width] * input[px + py * width];
            }
        }
        matrixDelta[x + y * width] = sum * ls;
    }
}


extern "C"
__global__ void matrixConvExtendBorderBackpropagationErrorTraversal(int width, int height, int mx, int my,
            double* in, double* matrix, double* out) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height) {
        int hx = mx >> 1;
        int hy = my >> 1;

        double sum = 0;
        for (int dy = 0; dy < mx; dy++) {
            int py = max(0, min(height - 1, y + my - 1 - dy - hy));

            for (int dx = 0; dx < mx; dx++) {
                int px = max(0, min(width - 1, x + mx - 1 - dx - hx));

                sum += in[px + py * width] * matrix[dx + dy * mx];
            }
        }
        out[x + y * width] = sum;
    }
}
extern "C"
__global__ void matrixConvExtendBorderBackpropagationWeightCorrection(int width, int height, int mx, int my, double ls,
            double* input, double* error, double* matrixDelta) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < mx && y < my) {
        int hx = mx >> 1;
        int hy = my >> 1;

        int sx = x - hx;
        int sy = y - hy;

        double sum = 0;
        for (int dy = 0; dy < height; dy++)
            for (int dx = 0; dx < width; dx++) {
                int px = max(0, min(width - 1, dx - sx));
                int py = max(0, min(height - 1, dy - sy));
                sum += error[dx + dy * width] * input[px + py * width];
            }
        matrixDelta[x + y * width] = sum * ls;
    }
}


extern "C"
__global__ void matrixConvRepeatBorderBackpropagationErrorTraversal(int width, int height, int mx, int my,
            double* in, double* matrix, double* out) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height) {
        int hx = mx >> 1;
        int hy = my >> 1;

        double sum = 0;
        for (int dy = 0; dy < mx; dy++) {
            int py = y + my - 1 - dy - hy;
            if (py < 0) py += height;
            if (py >= height) py -= height;

            for (int dx = 0; dx < mx; dx++) {
                int px = x + mx - 1 - dx - hx;
                if (px < 0) px += width;
                if (px >= width) px -= width;

                sum += in[px + py * width] * matrix[dx + dy * mx];
            }
        }
        out[x + y * width] = sum;
    }
}
extern "C"
__global__ void matrixConvRepeatBorderBackpropagationWeightCorrection(int width, int height, int mx, int my, double ls,
            double* input, double* error, double* matrixDelta) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < mx && y < my) {
        int hx = mx >> 1;
        int hy = my >> 1;

        int sx = x - hx;
        int sy = y - hy;

        double sum = 0;
        for (int dy = 0; dy < height; dy++) {
            int py = dy - sy;
            if (py < 0) py += height;
            if (py >= height) py -= height;
            for (int dx = 0; dx < width; dx++) {
                int px = dx - sx;
                if (px < 0) px += width;
                if (px >= width) px -= width;
                sum += error[dx + dy * width] * input[px + py * width];
            }
        }
        matrixDelta[x + y * width] = sum * ls;
    }
}


extern "C"
__global__ void maxPooling(int width, int height,
            int rate,
            double* in, double* out) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height) {
        int rx = x * rate;
        int ry = y * rate;
        int rw = width * rate;

        double val = -1e38;
        for (int dy = 0; dy < rate; dy++)
            for (int dx = 0; dx < rate; dx++)
                val = max(val, in[(rx + dx) + (ry + dy) * rw]);

        out[x + y * width] = val;
    }
}
extern "C"
__global__ void minPooling(int width, int height,
            int rate,
            double* in, double* out) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height) {
        int rx = x * rate;
        int ry = y * rate;
        int rw = width * rate;

        double val = 1e38;
        for (int dy = 0; dy < rate; dy++)
            for (int dx = 0; dx < rate; dx++)
                val = min(val, in[(rx + dx) + (ry + dy) * rw]);

        out[x + y * width] = val;
    }
}
extern "C"
__global__ void maxminPoolingBackpropagation(int width, int height, int rate,
            double* in, double* out, 
            double* errorNext, double* errorPrev) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height) {
        int rx = x / rate;
        int ry = y / rate;
        int rw = width / rate;

        errorPrev[x + y * width] = 0;
        if (abs(in[x + y * width] - out[rx + ry * rw]) < 1e-5) 
            errorPrev[x + y * width] = errorNext[rx + ry * width];
    }
}

extern "C"
__global__ void avgPooling(int width, int height,
            int rate,
            double* in, double* out) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height) {
        int rx = x * rate;
        int ry = y * rate;
        int rw = width * rate;

        double val = 0;
        for (int dy = 0; dy < rate; dy++)
            for (int dx = 0; dx < rate; dx++)
                val += in[(rx + dx) + (ry + dy) * rw];

        out[x + y * width] = val / (rate * rate);
    }
}
extern "C"
__global__ void avgPoolingBackpropagation(int width, int height, int rate,
            double* in, double* out, 
            double* errorNext, double* errorPrev) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height) {
        int rx = x / rate;
        int ry = y / rate;
        int rw = width / rate;

        errorPrev[x + y * width] = errorNext[rx + ry * width] * in[x + y + width] / (out[rx + ry * rw] * rate * rate);
    }
}